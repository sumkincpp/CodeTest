#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <math.h>


#define K 8
#define N pow(2, K)

typedef thrust::tuple<int, float>            tpl2int;
typedef thrust::device_vector<float>::iterator intiter;
typedef thrust::counting_iterator<int>     countiter;
typedef thrust::tuple<intiter, countiter>  tpl2intiter;
typedef thrust::zip_iterator<tpl2intiter>  idxzip;

__host__ __device__
int isbitset (int val, int bitnum)
{
  return (val & (1 << bitnum)) != 0;
}
__host__ __device__
int get_spin(int val, int spin_number) 
{
  if (isbitset(val, spin_number)) return 1;
  else return -1;
}

struct Ising2DStep : public thrust::unary_function<tpl2int, int>
{
  
  float _j1;
  float _j2;

  Ising2DStep(float j1, float j2) : _j1(j1), _j2(j2) {}

  __host__ __device__
  float operator()(const tpl2int& x) const
  {
    int idx = x.get<1>();
    float val = x.get<0>();

    return val + _j1*get_spin(idx, 0)*get_spin(idx, 1) + _j2*get_spin(idx, 0)*get_spin(idx, K);
  }
};

inline int gen_j () { return (rand() % 2) * 2 - 1; }

int main() 
{

  thrust::device_vector<float> source(N);
  //thrust::sequence(source.begin(), source.end());
  thrust::fill(source.begin(), source.end(), 0);

  thrust::device_vector<float> result(N);
  thrust::counting_iterator<int> idxfirst(0);
  thrust::counting_iterator<int> idxlast = idxfirst + N;

  for (int i = 0; i < 10000; ++i) {
    // We copy same numbers to the end of vector to reduce them later => 2*2^K
    source.insert(source.end(), source.begin(), source.end());

    idxzip first = thrust::make_zip_iterator(thrust::make_tuple(source.begin(), idxfirst));
    idxzip  last = thrust::make_zip_iterator(thrust::make_tuple(source.end(), idxlast));

    int j2 = gen_j();

    //if (i % K == 0) j2 = 0;

    Ising2DStep isingStep(gen_j(), j2);

    // Here we are getting 2*2^K numbers that should reduced to 2^K
    thrust::transform(first, last, result.begin(), isingStep);

    source.clear();

    // Following the pattern -> we find minimum of pairs
    for (thrust::device_vector<float>::iterator it = result.begin(); it != result.end(); it += 2) {
      source.push_back (min( *it, *(it+1)));
    }

    /*for (thrust::device_vector<float>::iterator it = source.begin(); it != source.end(); it += 1) {
      std::cout << *it << " ";
    }
    std::cout << std::endl;*/

    //if ( i == 10) break;
  }
  // Printing values from source
  thrust::copy(source.begin(), source.end(), std::ostream_iterator<float>( std::cout, " "));


  return 0;

}